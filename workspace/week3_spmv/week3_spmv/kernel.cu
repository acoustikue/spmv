#include "hip/hip_runtime.h"
﻿// Week 3
// cuSPARSE Example.
// acoustikue@yonsei.ac.kr
// written by SukJoon Oh

#include "hip/hip_runtime.h"
#include ""
#include <hipsparse.h>	// hipsparseSpMV

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "mmio.h"


#define CASE4
//#define CUSPARSE
#define CUSPARSE_CSR
// #define KERNEL_CSR_SCALAR
#define KERNEL_CSR_VECTOR

#ifdef CASE1
#define M	10
#define N	10
#define NZ	20
#define MTX_FILE	"10_10_sample_mat.mtx"
#endif
#ifdef CASE2
#define M	1024
#define N	1024
#define NZ	209715
#define MTX_FILE	"1024_1024_sample_mat.mtx"
#endif
#ifdef CASE3
#define M	2048
#define N	2048
#define NZ	838861
#define MTX_FILE	"2048_2048_sample_mat.mtx"
#endif
#ifdef CASE4
#define M	4096
#define N	4096
#define NZ	3355443
#define MTX_FILE	"4096_4096_sample_mat.mtx"
#endif

#define CUDA_ERR(func)                                                         \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CUSPARSE_ERR(func)                                                     \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}


// Author: SukJoon Oh
// acoustikue@yonsei.ac.kr
// Reads MM file.
void read_matrix(int* argJR, int* argJC, float* argAA) {

	int m = M;
	int n = N;
	int nz = NZ;

	FILE* MTX;
	MTX = fopen(MTX_FILE, "r");
	 
	MM_typecode matrix_code;

	// Read banner, type, etc essential infos
	// Verification steps are ignored.
	if (mm_read_banner(MTX, &matrix_code) != 0) exit(1);
	mm_read_mtx_crd_size(MTX, &m, &n, &nz); // Over max 1025

	printf("Market Market type: [%s]\n", mm_typecode_to_str(matrix_code));

	// COO format
	for (register int i = 0; i < NZ; i++)
		fscanf_s(MTX, "%d %d %f\n", &argJR[i], &argJC[i], &argAA[i]);

	fclose(MTX);
}


// 
// CSR scalar kernel function
__global__ void ker_csr_spmv_scalar(
	const int* argJR, const int* argJC, const float* argAA,
	const float* arg_x, float* arg_y) {

	int idx		= blockDim.x * blockIdx.x + threadIdx.x;
	float sum	= 0;

	for (int i = argJR[idx] - 1; i < argJR[idx + 1] - 1; i++)
		sum		+= (argAA[i] * arg_x[argJC[i] - 1]);

	arg_y[idx]	+= sum;
};




//
// CSR vector kernel function
__global__ void ker_csr_spmv_vector(
	const int* argJR, const int* argJC, const float* argAA,
	const float* arg_x, float* arg_y) {

	// Thread : 32 * M

	int tid		= blockDim.x * blockIdx.x + threadIdx.x;
	int wid		= tid / 32;
	int lidx	= tid & 31;
	float sum	= 0;

	for (int i = argJR[wid] - 1 + lidx; i < argJR[wid + 1] - 1; i += 32)
		sum += argAA[i] * arg_x[argJC[i] - 1];

	for (int i = 16; i > 0; i /= 2)
		sum += __shfl_down_sync(0xFFFFFFFF, sum, i);

	if (lidx == 0) arg_y[wid] = sum;
};




// ---- main() ----
// Entry
int main()
{
	//
	// ---- Step 1. Load info ----
	int* host_JR	= (int*)malloc(NZ * sizeof(int));
	int* host_JC	= (int*)malloc(NZ * sizeof(int));
	float* host_AA	= (float*)malloc(NZ * sizeof(float));
	int* host_P		= (int*)malloc(NZ * sizeof(int));

	read_matrix(host_JR, host_JC, host_AA); // prepare elements

	//
	// ---- Step 2. Handle create, bind a stream ---- 
	int* device_JR			= NULL;
	int* device_JC			= NULL;
	float* device_AA		= NULL;
	float* device_AA_sorted	= NULL;
	int* device_P			= NULL;

	void* buffer			= NULL;
	size_t buffer_size		= 0;

	hipsparseHandle_t handle = NULL;
	hipStream_t stream		= NULL;

	CUDA_ERR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUSPARSE_ERR(hipsparseCreate(&handle));
	CUSPARSE_ERR(hipsparseSetStream(handle, stream));

	//
	// ---- Step 3. Allocate Buffer ---- 
	CUSPARSE_ERR(
		hipsparseXcoosort_bufferSizeExt(
			handle,
			M, N, NZ,
			device_JR, device_JC, &buffer_size
		)
	);

	CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_P, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_AA, sizeof(float) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_AA_sorted, sizeof(float) * NZ));
	CUDA_ERR(hipMalloc((void**)&buffer, sizeof(char) * buffer_size));

	CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipMemcpy(device_AA, host_AA, sizeof(float) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipDeviceSynchronize());

	//
	// ---- Step 4. Setup permutation vector P to Identity ---- 
	CUSPARSE_ERR(hipsparseCreateIdentityPermutation(handle, NZ, device_P));

	//
	// ---- Step 5. Sort ---- 
	CUSPARSE_ERR(
		hipsparseXcoosortByRow(handle, M, N, NZ, device_JR, device_JC, device_P, buffer)
	);

	// Gather
	// CUSPARSE_ERR(hipsparseDgthr(
	//	handle, NZ, device_AA, device_AA_sorted, device_P, HIPSPARSE_INDEX_BASE_ZERO));
	CUSPARSE_ERR(hipsparseSgthr(
		handle, NZ, device_AA, device_AA_sorted, device_P, HIPSPARSE_INDEX_BASE_ZERO));
	CUDA_ERR(hipDeviceSynchronize());

	// Fetch back
	CUDA_ERR(hipMemcpy(host_JR, device_JR, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_JC, device_JC, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_P, device_P, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_AA, device_AA_sorted, sizeof(float) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipDeviceSynchronize());

	// ---- Step 6. Free resources ---- 
#ifdef CUSPARSE_CSR
	if (device_JR) hipFree(device_JR);
	if (device_JC) hipFree(device_JC);
#endif
	if (device_P) hipFree(device_P);
	if (device_AA) hipFree(device_AA);
	if (buffer) hipFree(buffer);
	if (handle) hipsparseDestroy(handle);
	if (stream) hipStreamDestroy(stream);

	free(host_P); // Unnecessary


#if defined( CUSPARSE_CSR )
	int* t_JR	= (int*)calloc((M + 1), sizeof(int));
	int* t_JC	= (int*)malloc(NZ * sizeof(int));
	float* t_AA = (float*)malloc(NZ * sizeof(float));
	for (int i = 0; i < M + 1; i++) t_JR[i]++;

	for (int i = 0; i < NZ; i++) {
		t_AA[i] = host_AA[i];
		t_JC[i] = host_JC[i];
		t_JR[host_JR[i]]++;
	}

	for (int i = 0; i < M; i++)	t_JR[i + 1] += (t_JR[i] - 1);

	free(host_JR);
	free(host_JC);
	free(host_AA);

	host_JR = t_JR;
	host_JC = t_JC;
	host_AA = t_AA;

#endif


	// ----               ----
	// ---- cuSPARSE SpMV ----
	// ----               ----
	handle		= NULL;
	buffer		= NULL;
	buffer_size = 0;

	float elapsed = 0;
	hipEvent_t start, stop;

#ifdef CUSPARSE
	{ // SpMV
		printf("\n#### \tSpMV cuSPARSE \t####\n");
		// ---- Step 7. Define variables
		const float alpha	= 1;
		const float beta	= 0;

		float host_y[N]		= {0, };
		float host_x[M];

		float* device_x		= NULL;
		float* device_y		= NULL;

		for (auto& elem : host_x) elem = 1;

		hipsparseSpMatDescr_t sp_mtx; // device
		hipsparseDnVecDescr_t dn_x, dn_y; // device

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start); // Timer start

		// ---- Step 8. Get your GPU memory ready ----
		CUDA_ERR(hipMalloc((void**)&device_x, sizeof(float) * M));
		CUDA_ERR(hipMalloc((void**)&device_y, sizeof(float) * N));

		CUDA_ERR(hipMemcpy(device_x, host_x, sizeof(float) * M, hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_y, host_y, sizeof(float) * N, hipMemcpyHostToDevice));

#ifdef CUSPARSE_CSR
		CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * (M + 1)));
		CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));

		CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * (M + 1), hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
#endif

		CUSPARSE_ERR(hipsparseCreate(&handle));

		// Create sparse matrix
#ifndef CUSPARSE_CSR
		CUSPARSE_ERR(
			hipsparseCreateCoo(
				&sp_mtx, 
				M, N, NZ, device_JR, device_JC, device_AA_sorted,
				HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F)
		);
#else
		CUSPARSE_ERR(
			hipsparseCreateCsr(
				&sp_mtx,
				M, N, NZ, device_JR, device_JC, device_AA_sorted,
				HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F)
		);
#endif
		
		CUSPARSE_ERR(hipsparseCreateDnVec(&dn_x, N, device_x, HIP_R_32F));
		CUSPARSE_ERR(hipsparseCreateDnVec(&dn_y, M, device_y, HIP_R_32F));

#ifndef CUSPARSE_CSR
		CUSPARSE_ERR(hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_COOMV_ALG, &buffer_size));
#else
		CUSPARSE_ERR(hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1, &buffer_size));
#endif

		CUDA_ERR(hipMalloc(&buffer, buffer_size));
		
		// ---- Step 9. Do SpMV ----
#ifndef CUSPARSE_CSR
		CUSPARSE_ERR(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_COOMV_ALG, buffer));
#else
		CUSPARSE_ERR(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1, buffer));


		// ---- Step 11. Destroy ----
		CUSPARSE_ERR(hipsparseDestroySpMat(sp_mtx));
		CUSPARSE_ERR(hipsparseDestroyDnVec(dn_x));
		CUSPARSE_ERR(hipsparseDestroyDnVec(dn_y));
#endif

		// ---- Step 10. Fetch the result ----
		CUDA_ERR(hipMemcpy(host_y, device_y, N * sizeof(float), hipMemcpyDeviceToHost));

		// Record
		hipEventRecord(stop); // timer end
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);

		for (int i = 0; i < 10; i++) 
			printf("%7.1f", host_y[i]); // Check		

		// ---- Step 12. Return resources ----
		if (device_JR) hipFree(device_JR);
		if (device_JC) hipFree(device_JC);
		if (device_AA_sorted) hipFree(device_AA_sorted);
		if (device_x) hipFree(device_x);
		if (device_y) hipFree(device_y);
		if (buffer) hipFree(buffer);
		if (handle) hipsparseDestroy(handle);


		printf("\nElapsed: %fms\n", elapsed);

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}	
#else

	// ----             ----
	// ---- Kernel SpMV ----
	// ----             ----
	{
		printf("\n#### \tSpMV Kernel \t####\n");
		float host_y[N]		= { 0, };
		float host_x[M];

		float* device_x		= NULL;
		float* device_y		= NULL;

		for (auto& elem : host_x) elem = 1;

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start); // Timer start

		CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * (M + 1)));
		CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));
		CUDA_ERR(hipMalloc((void**)&device_x, sizeof(float) * M));
		CUDA_ERR(hipMalloc((void**)&device_y, sizeof(float) * N));

		CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * (M + 1), hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_x, host_x, sizeof(float) * M, hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_y, host_y, sizeof(float) * N, hipMemcpyHostToDevice));

#ifdef KERNEL_CSR_SCALAR
#ifdef CASE1
		ker_csr_spmv_scalar<<<1, M>>>(
#endif
#ifdef CASE2
		ker_csr_spmv_scalar<<<1, M>>>(
#endif
#ifdef CASE3
		ker_csr_spmv_scalar<<<2, M / 2>>>(
#endif
#ifdef CASE4
		ker_csr_spmv_scalar<<<4, M / 4>>>(
#endif
				device_JR, device_JC, device_AA_sorted, device_x, device_y
			);
		// hipDeviceSynchronize();
#endif
#ifdef KERNEL_CSR_VECTOR
#ifdef CASE1
		ker_csr_spmv_vector <<<1, 32 * M>>>(
#endif
#ifdef CASE2
		ker_csr_spmv_vector <<<32, M>>>(
#endif
#ifdef CASE3
		ker_csr_spmv_vector <<<64, 1024>>>(
#endif
#ifdef CASE4
		ker_csr_spmv_vector <<<128, 1024>>>(
#endif
			device_JR, device_JC, device_AA_sorted, device_x, device_y
		);
#endif

		// ---- Step 10. Fetch the result ----
		CUDA_ERR(hipMemcpy(host_y, device_y, N * sizeof(float), hipMemcpyDeviceToHost));

		// Record
		hipEventRecord(stop); // timer end
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);

		for (int i = 0; i < 10; i++)
			printf("%7.1f", host_y[i]); // Check		

		if (device_JR) hipFree(device_JR);
		if (device_JC) hipFree(device_JC);
		if (device_AA_sorted) hipFree(device_AA_sorted);
		if (device_x) hipFree(device_x);
		if (device_y) hipFree(device_y);
		if (buffer) hipFree(buffer);
		if (handle) hipsparseDestroy(handle);

		printf("\nElapsed: %fms\n", elapsed);

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

#endif

	free(host_JR);
	free(host_JC);
	free(host_AA);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	if (hipDeviceReset() != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}